#include "hip/hip_runtime.h"
#include <iostream>
#include "dnn.hpp"

#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

using namespace std;


// this will be called in main() once we have tiling values figured out
#ifndef Tii
 // Tiling Sizes
 #define Tnn 32
 #define Tii 32
 //#define Tn 5
 //#define Ti 25
 #define Tn 16
 #define Ti 16
#endif

void fill_classifier(VTYPE (&weights)[Nn][Ni], VTYPE (&data_in)[Ni],
   VTYPE (&data_out)[Nn], VTYPE (&data_out_block)[Nn]) {

  for(int n = 0; n < Nn; ++n) {
    for(int i = 0; i < Ni; ++i) {
      //weights[n][i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
      weights[n][i] = static_cast <float> (i*n);
    }
  }
  for(int i = 0; i < Ni; ++i) {
    //data_in[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
    data_in[i] = 1.0f;
  }
  for(int n = 0; n < Nn; ++n) {
    data_out[n] = 0;
    data_out_block[n] = 0;
  }
}

__global__ void classifier_layer_gpu(VTYPE *d_weights, VTYPE *d_data_in, VTYPE *d_data_out) {
  // blockDim = threads in block
  // 1 thread per output data
  // printf("Kernel called from block %d, thread %d\n", blockIdx.x, threadIdx.x);
  int ix = blockIdx.x * blockDim.x + threadIdx.x;

  VTYPE tmp = 0;
  if(ix < Nn){
    for (int n = 0; n < Ni; n++) {
      int startidx = ix * Ni;
      tmp += d_weights[startidx + n] * d_data_in[n];
    }
    d_data_out[ix] = tmp;
  }
}

// original host version of classifier layers
void classifier_layer_host(VTYPE (&weights)[Nn][Ni], VTYPE (&data_in)[Ni], VTYPE (&data_out)[Nn]) {
  for (int n = 0; n < Nn; n++) {
    VTYPE tmp=0;
    for (int i = 0; i < Ni; i++) {
      tmp += weights[n][i] * data_in[i];
    }
    data_out[n] = transfer(tmp);
  }
}

// not yet converted to CUDA operation
void classifier_layer_blocked_host(VTYPE (&weights)[Nn][Ni], VTYPE (&data_in)[Ni],
                             VTYPE (&data_out_block)[Nn]) {
 VTYPE sum[Nn]={0};
 for (int nnn = 0; nnn < Nn; nnn += Tnn) { // tiling for output neurons;
   for (int iii = 0; iii < Ni; iii += Tii) { // tiling for input neurons;
     for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
       for (int ii = iii; ii < iii + Tii; ii += Ti) {
         // — Original code —
         for (int n = nn; n < nn + Tn; n++) {
           VTYPE sum_sc=0;
           for (int i = ii; i < ii + Ti; i++) {
             sum_sc += (weights[n][i] * data_in[i]);
           }
           sum[n]+=sum_sc;
         }
       }
     }
   }
   for (int nn = nnn; nn < nnn + Tnn; nn++) {
     data_out_block[nn] = transfer(sum[nn]);
   }
 }
}

//Arrays:
VTYPE weights[Nn][Ni] __attribute__((aligned(64)));
VTYPE data_in[Ni] __attribute__((aligned(64)));
VTYPE data_out[Nn] __attribute__((aligned(64)));
VTYPE data_out_block[Nn] __attribute__((aligned(64)));
VTYPE data_out_gpu[Nn] __attribute__((aligned(64)));

int main(int argc, char** argv) {


  cout << "initializing arrays\n";
  //fill_classifier(weights,data_in,data_out,data_out_block);
  fill_classifier(weights,data_in,data_out,data_out_block);

  cout << "Host classifier computation begin\n";
  begin_roi();
  classifier_layer_host(weights,data_in,data_out);
  end_roi();
  cout << "Host classifier computation end\n";

  cout << "blocked computation begin!\n";
  //begin_roi();
  //classifier_layer_blocked_host(weights,data_in,data_out_block);
  //end_roi();
  cout << "blocked computation complete!\n";

  // allocate arrays in device memory
  int inputSize = sizeof(VTYPE)*Ni;
  int weightsSize = sizeof(VTYPE)*Ni*Nn;
  int outputSize = sizeof(VTYPE)*Nn;
  VTYPE *d_data_in, *d_weights, *d_data_out;
  hipMalloc(&d_data_in, inputSize);
  hipMalloc(&d_weights, weightsSize);
  hipMalloc(&d_data_out, outputSize);

  // transfer data to device
  hipMemcpy(d_data_in, &data_in, inputSize, hipMemcpyHostToDevice);
  hipMemcpy(d_weights, &weights, weightsSize, hipMemcpyHostToDevice);
  hipMemcpy(d_data_out, &data_out, outputSize, hipMemcpyHostToDevice);

  int threadsPerBlock = 256; // threads per block
  int numBlocks = (Nn + (threadsPerBlock - 1)) / threadsPerBlock; // number of blocks

  cout << "Cuda classifier computation begin\n";
  begin_roi();
  classifier_layer_gpu<<<numBlocks,threadsPerBlock>>>(d_weights,d_data_in,d_data_out);
  hipDeviceSynchronize();
  hipMemcpy(&data_out_gpu, d_data_out, outputSize, hipMemcpyDeviceToHost);
  end_roi();
  cout << "Cuda classifier computation done\n";

  compare(data_out, data_out_gpu, Nn);

  hipFree(d_data_in);
  hipFree(d_data_out);
  hipFree(d_weights);

  cout << "done\n";
}
